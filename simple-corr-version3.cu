
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include <math.h>
int64_t getnumlines(const char *fname,const char comment);
int64_t read_ascii_file(const char *filename,double **xpos,double **ypos,double **zpos);
int setup_bins_double(const char *fname,double *rmin,double *rmax,int *nbin,double **rupp);

int64_t getnumlines(const char *fname,const char comment)
{
  //FILE *fp = ((void *)0);
  FILE *fp = NULL;
  const int MAXLINESIZE = 10000;
  int64_t nlines = 0;
  char str_line[MAXLINESIZE];
  fp = fopen(fname,"rt");
  //if (fp == ((void *)0)) {
  if (fp == NULL) {
    fprintf(stderr,"Error: Could not open file `%s'\n",fname);
    //perror(((void *)0));
    perror(NULL);
    return (- 1);
  }
  while(1){
    //if (fgets(str_line,MAXLINESIZE,fp) != ((void *)0)) {
//WARNING: this does not remove white-space. You might
//want to implement that (was never an issue for me)
    if (fgets(str_line,MAXLINESIZE,fp) != (NULL)) {
      if (str_line[0] != comment) 
        nlines++;
    }
     else 
      break; 
  }
  fclose(fp);
  return nlines;
}

int64_t read_ascii_file(const char *filename,double **xpos,double **ypos,double **zpos)
{
  int64_t numlines = getnumlines(filename,'#');
  if (numlines <= 0) 
    return numlines;
    //Ritu: besides change (void *) to NULL, typecasting in the three lines below
  double *x = (double *) (calloc(numlines,sizeof(( *x))));
  double *y = (double *) (calloc(numlines,sizeof(( *y))));
  double *z = (double *) (calloc(numlines,sizeof(( *z))));
  //if (x == ((void *)0) || y == ((void *)0) || z == ((void *)0)) {
  if (x == NULL || y == NULL || z == NULL) {
    free(x);
    free(y);
    free(z);
    fprintf(stderr,"Error: Could not allocate memory for %ld elements for the (x/y/z) arrays\n",numlines);
    //perror(((void *)0));
    perror(NULL);
    return (- 1);
  }
  FILE *fp = fopen(filename,"rt");
  //if (fp == ((void *)0)) {
  if (fp == NULL) {
    fprintf(stderr,"Error:Could not open file `%s' in function %s\n",filename,__FUNCTION__);
    fprintf(stderr,"This is strange because the function `getnumlines' successfully counted the number of lines in that file\n");
    fprintf(stderr,"Did that file (`%s') just get deleted?\n",filename);
    //perror(((void *)0));
    perror(NULL);
    return (- 1);
  }
  int64_t index = 0;
  const int nitems = 3;
  const int MAXLINESIZE = 10000;
  char buf[MAXLINESIZE];
  while(1){
    if (fgets(buf,MAXLINESIZE,fp) != ((void *)0)) {
      int nread = sscanf(buf,"%lf %lf %lf",&x[index],&y[index],&z[index]);
      if (nread == nitems) {
        index++;
      }
    }
     else {
      break; 
    }
  }
  fclose(fp);
  if (index != numlines) {
    fprintf(stderr,"Error: There are supposed to be `%'ld lines of data in the file\n",numlines);
    fprintf(stderr,"But could only parse `%'ld lines containing (x y z) data\n",index);
    fprintf(stderr,"exiting...\n");
    return (- 1);
  }
   *xpos = x;
   *ypos = y;
   *zpos = z;
  return numlines;
}

int setup_bins_double(const char *fname,double *rmin,double *rmax,int *nbin,double **rupp)
{
//set up the bins according to the binned data file
//the form of the data file should be <rlow  rhigh ....>
  const int MAXBUFSIZE = 1000;
  char buf[MAXBUFSIZE];
  //FILE *fp = ((void *)0);
  FILE *fp = NULL;
  double low;
  double hi;
  const char comment = '#';
  const int nitems = 2;
  int nread = 0;
   *nbin = ((int )(getnumlines(fname,comment))) + 1;
  // *rupp = (calloc(( *nbin + 1),sizeof(double )));
  *rupp = (double *)(calloc(( *nbin + 1),sizeof(double )));
  if (rupp == ((void *)0)) {
    fprintf(stderr,"Error: Could not allocate memory for %d bins to store the histogram limits\n", *nbin + 1);
    //perror(((void *)0));
    perror(NULL);
    return 1;
  }
  fp = fopen(fname,"rt");
  if (fp == ((void *)0)) {
    free(( *rupp));
    fprintf(stderr,"Error: Could not open file `%s'..exiting\n",fname);
    //perror(((void *)0));
    perror(NULL);
    return 1;
  }
  int index = 1;
  while(1){
    if (fgets(buf,MAXBUFSIZE,fp) != ((void *)0)) {
      nread = sscanf(buf,"%lf %lf",&low,&hi);
      if (nread == nitems) {
        if (index == 1) {
           *rmin = low;
          ( *rupp)[0] = low;
        }
        ( *rupp)[index] = hi;
        index++;
      }
    }
     else {
      break; 
    }
  }
   *rmax = ( *rupp)[index - 1];
  fclose(fp);
  ( *rupp)[ *nbin] =  *rmax;
  ( *rupp)[ *nbin - 1] =  *rmax;
  return 0;
}




void __global__ kernel0(double * xpos,double * ypos,double * zpos,int64_t* npairs, double* rpavg,double * rupp,int64_t Npart,const double sqr_rmin,const double sqr_rmax,const int logbins,const double log10rmin,const double inv_dlogr,int nbins, int device_M , int device_N, int64_t* totalNpairs,double* totalRpavg){

 int64_t i =  blockIdx.x * blockDim.x + threadIdx.x;

 if(i< Npart) {
  for(int64_t j = (i+1);j < Npart;j++) {
      const double dx = xpos[i] - xpos[j];
      const double dy = ypos[i] - ypos[j];
      const double dz = zpos[i] - zpos[j];
      const double r2 = dx * dx + dy * dy + dz * dz;
      if(r2 < sqr_rmin || r2 >= sqr_rmax) 
         continue;
      const double r = sqrt(r2);
      if(logbins) {
        const int kbin =(int )((log10(r) - log10rmin) * inv_dlogr);
        totalNpairs[(i*nbins)+kbin + 1]++;
        totalRpavg[(i*nbins)+kbin + 1] += r;
      }else {
        for(int kbin = nbins - 1;kbin >= 1;kbin--) {
            if(r >= rupp[kbin - 1]) {
                totalNpairs[(i*nbins)+kbin]++;
                totalRpavg[(i*nbins)+kbin] += r;
                break;
            }
        }
      }
  }

 }
}

void __global__ kernel1(int64_t *npairs, double* rpavg,int nbins, int64_t Npart,int64_t* totalNpairs,double* totalRpavg){

 int64_t i =  blockIdx.x * blockDim.x + threadIdx.x;


 if(i==0){
     for(int64_t k=0; k<Npart; k++){
      for (int64_t j=0; j<nbins; j++){
              npairs[j] += totalNpairs[(k*nbins)+ j];
              rpavg[j] += totalRpavg[(k*nbins)+ j];
      }
     }

 }

}




int main(int argc,char **argv)
{
  double *device_rupp;
  double *device_rpavg;
  int64_t *device_npairs;
  double *device_zpos;
  double *device_ypos;
  double *device_xpos;
  int64_t *device_totalNpairs;
  double *device_totalRpavg;
  if (argc < 3) {
    fprintf(stderr,"\n\tUsage: %s `filename (string)' `filename-with-bins (string)' `[log bins (boolean)]'\n\n",argv[0]);
    fprintf(stderr,"\t************************************************************\n");
    fprintf(stderr,"\tRequired\n");
    fprintf(stderr,"\t--------\n");
    fprintf(stderr,"\t filename                string, an ascii file containing particle data (white-space-separated, 3 columns of x y z)\n");
    fprintf(stderr,"\t filename-with-bins      string, an ascii file containing <rlow rmax> specifying logarithmic bins (number of lines equal the number of bins)\n");
    fprintf(stderr,"\n\tOptional\n");
    fprintf(stderr,"\t--------\n");
    fprintf(stderr,"\t log-bins                boolean, default 0. Supply `1' indicating that the supplied bins are logarithmic (assumed to be log10)\n");
    fprintf(stderr,"\t************************************************************\n\n");
    return 1;
  }
  int logbins = 0;
  if (argc > 3) {
    logbins = atoi(argv[3]);
    fprintf(stderr,"Assuming that bins are logarithmic. Using logbins = %d\n",logbins);
  }
  double *xpos;
  double *ypos;
  double *zpos;
  int64_t Npart = read_ascii_file(argv[1],&xpos,&ypos,&zpos);
  if (Npart <= 0) {
    return Npart;
  }
  double rmin;
  double rmax;
  double *rupp;
  int nbins;
  int status = setup_bins_double(argv[2],&rmin,&rmax,&nbins,&rupp);
  if (status < 0) {
    return status;
  }
  //double *rpavg = (calloc(nbins,sizeof(( *rpavg))));
  //int64_t *npairs = (calloc(nbins,sizeof(( *npairs))));

  double *rpavg = (double *) (calloc(nbins,sizeof(( *rpavg))));
  int64_t *npairs = (int64_t *) (calloc(nbins,sizeof(( *npairs))));

  double *totalRpavg = (double *) (calloc((nbins*Npart),sizeof(( *rpavg))));
  int64_t *totalNpairs = (int64_t *) (calloc((nbins*Npart),sizeof(( *npairs))));

  const double sqr_rmin = rmin * rmin;
  const double sqr_rmax = rmax * rmax;
  const double log10rmin = log10(rmin);
  const double log10rmax = log10(rmax);
/* because of the way nbins is implemented
       bin `0' is underflow, and bin `nbin' is overflow  */
  const double dlogr = (log10rmax - log10rmin) / (nbins - 1);
  const double inv_dlogr = 1.0 / dlogr;

//Please note this is the section wherein the number of blocks and threads are calculated.  To change the number of threads alter the dimBlock whereas to change the number of blocks alter the dimGrid

int D_rows = (Npart > 1024 ) ? Npart/1024 : Npart;
int D_cols = (Npart > 1024 ) ? 1024 : 1;
//Ritu:updating the number of D_rows
if ( Npart % 1024){
 D_rows++;
}

//printf("\nD_rows:%d, D_cols:%d\n",D_rows, D_cols);

dim3 dimGrid(D_rows,1);
dim3 dimBlock(D_cols,1);
hipMalloc((void **) &device_xpos,(Npart)*sizeof(int64_t));
hipMemcpy(device_xpos,xpos,(Npart)*sizeof(double),hipMemcpyHostToDevice);
hipMalloc((void **) &device_ypos,(Npart)*sizeof(double));
hipMemcpy(device_ypos,ypos,(Npart)*sizeof(double),hipMemcpyHostToDevice);
hipMalloc((void **) &device_zpos,(Npart)*sizeof(double));
hipMemcpy(device_zpos,zpos,(Npart)*sizeof(double),hipMemcpyHostToDevice);

for (int64_t j=0; j<nbins; j++){
    npairs[j]  = 0;
    rpavg[j] = 0.0;
}
hipMalloc((void **) &device_npairs,(nbins)*sizeof(int64_t));
hipMemcpy(device_npairs,npairs,(nbins)*sizeof(int64_t),hipMemcpyHostToDevice);

hipMalloc((void **) &device_rpavg,(nbins)*sizeof(double));
hipMemcpy(device_rpavg,rpavg,(nbins)*sizeof(double),hipMemcpyHostToDevice);

hipMalloc((void **) &device_rupp,(nbins)*sizeof(double));
hipMemcpy(device_rupp,rupp,(nbins)*sizeof(double),hipMemcpyHostToDevice);

hipMalloc((void **) &device_totalNpairs,(nbins*Npart)*sizeof(int64_t));

hipMalloc((void **) &device_totalRpavg,(nbins*Npart)*sizeof(double));

for (int64_t j=0; j<nbins*Npart; j++){
    totalNpairs[j]  = 0;
    totalRpavg[j] = 0.0;
}

hipMemcpy(device_totalNpairs,totalNpairs,(nbins*Npart)*sizeof(int64_t),hipMemcpyHostToDevice);
hipMemcpy(device_totalRpavg,totalNpairs,(nbins*Npart)*sizeof(double),hipMemcpyHostToDevice);

//kernel0<<<dimGrid,dimBlock>>>(device_xpos,device_ypos,device_zpos,device_rupp,Npart,sqr_rmin,sqr_rmax,logbins,log10rmin,inv_dlogr,nbins,1,Npart, device_totalNpairs,device_totalRpavg);

kernel0<<<dimGrid,dimBlock>>>(device_xpos,device_ypos,device_zpos,device_npairs,device_rpavg,device_rupp,Npart,sqr_rmin,sqr_rmax,logbins,log10rmin,inv_dlogr,nbins,1,Npart, device_totalNpairs,device_totalRpavg);


//size_t sharedMemory = (Npart*nbins*sizeof(int64_t)) + (Npart*nbins*sizeof(double));

//kernel0<<<dimGrid,dimBlock, sharedMemory>>>(device_xpos,device_ypos,device_zpos,device_npairs,device_rpavg,device_rupp,Npart,sqr_rmin,sqr_rmax,logbins,log10rmin,inv_dlogr,nbins,totalNpairs,totalRpavg);
/*
  int IPT_function_replace;
*/

hipDeviceSynchronize();

hipFree(device_xpos);
hipFree(device_ypos);
hipFree(device_zpos);

kernel1<<<dimGrid,dimBlock>>>(device_npairs,device_rpavg,nbins,Npart,device_totalNpairs,device_totalRpavg);

hipMemcpy(npairs,device_npairs,(nbins)*sizeof(int64_t), hipMemcpyDeviceToHost);

hipMemcpy(rpavg,device_rpavg,(nbins)*sizeof(double), hipMemcpyDeviceToHost);

hipFree(device_npairs);
hipFree(device_rpavg);


 //cudaMemcpy(totalNpairs,device_totalNpairs,(nbins*Npart)*sizeof(int64_t), cudaMemcpyDeviceToHost);
 hipFree(device_totalNpairs);

 //cudaMemcpy(totalRpavg,device_totalRpavg,(nbins*Npart)*sizeof(double), cudaMemcpyDeviceToHost);
 hipFree(device_totalRpavg);

  hipFree(device_rupp);
  double rlow = rupp[0];
  for (int i = 1; i < nbins; i++) {
    if (npairs[i] > 0) 
      rpavg[i] /= npairs[i];
    fprintf(stdout,"%e\t%e\t%e\t%12lu\t%e\n",rlow,rupp[i],rpavg[i],npairs[i],0.0);
    rlow = rupp[i];
  }

  free(xpos);
  free(ypos);
  free(zpos);
  free(rupp);
  free(npairs);
  free(rpavg);
  free(totalNpairs);
  free(totalRpavg);
  return 0;
}

